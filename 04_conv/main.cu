#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <iostream>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <opencv2/opencv.hpp>
#include <chrono>
#include "tools.h"
#include "kernel.h"

using namespace std;
using namespace cv;

int main()
{
    // 创建 CUDA 事件
    hipError_t err;
    hipEvent_t gpu_start, gpu_stop;
    err = hipEventCreate(&gpu_start);
    if (err != hipSuccess)
    {
        printf("Error creating start event: %s\n", hipGetErrorString(err));
        return -1;
    }
    err = hipEventCreate(&gpu_stop);
    if (err != hipSuccess)
    {
        printf("Error creating stop event: %s\n", hipGetErrorString(err));
        return -1;
    }

    // 读取原始图像
    int img_rows = 2048;
    int img_cols = 2048;
    int img_size = img_rows * img_cols;
    float *h_img = new float[img_size];
    float *d_img;
    hipMalloc(&d_img, img_size * sizeof(float));
    const char *img_filename = "data/img.dat";
    readFile(img_filename, h_img, img_size);
    hipMemcpy(d_img, h_img, img_size * sizeof(float), hipMemcpyHostToDevice);
    // showImg(h_img, img_rows, img_cols, "Original Image", 512, 512);

    // 生成高斯滤波器
    int mask_width = 19;
    float *h_mask = new float[mask_width * mask_width];
    float *d_mask;
    hipMalloc(&d_mask, mask_width * mask_width * sizeof(float));
    generateGaussFilter(h_mask, mask_width);
    hipMemcpy(d_mask, h_mask, mask_width * mask_width * sizeof(float), hipMemcpyHostToDevice);

    // CPU 卷积
    // 获取起始时间
    auto cpu_start = std::chrono::high_resolution_clock::now();
    float *h_img_blur_cpu = new float[img_size];
    convCPU(h_img, h_mask, h_img_blur_cpu, img_rows, img_cols, mask_width);
    // 获取结束时间
    auto cpu_stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> cpu_elapsed = cpu_stop - cpu_start;
    cout << "CPU Time: " << cpu_elapsed.count() << "s" << endl;
    showImg(h_img_blur_cpu, img_rows, img_cols, "Blurred Image CPU", 512, 512);

    float *h_img_blur_gpu = new float[img_size];
    float *d_img_blur_gpu;
    hipMalloc(&d_img_blur_gpu, img_size * sizeof(float));
    
    dim3 block(16, 16);
    dim3 grid((img_cols + block.x - 1) / block.x, (img_rows + block.y - 1) / block.y);
    // 记录起始事件
    err = hipEventRecord(gpu_start, 0);
    if (err != hipSuccess)
    {
        printf("Error recording start event: %s\n", hipGetErrorString(err));
        return -1;
    }
    convGPUSharedMem<<<grid, block>>>(d_img, d_mask, d_img_blur_gpu, img_rows, img_cols);
    // convGPUGlobalMem<<<grid, block>>>(d_img, d_mask, d_img_blur_gpu, img_rows, img_cols);
    // 记录结束事件
    err = hipEventRecord(gpu_stop, 0);
    if (err != hipSuccess)
    {
        printf("Error recording stop event: %s\n", hipGetErrorString(err));
        return -1;
    }
    // 等待事件完成
    err = hipEventSynchronize(gpu_stop);
    if (err != hipSuccess)
    {
        printf("Error synchronizing stop event: %s\n", hipGetErrorString(err));
        return -1;
    }
    // 计算时间差
    float gpu_elapsed;
    err = hipEventElapsedTime(&gpu_elapsed, gpu_start, gpu_stop);
    if (err != hipSuccess)
    {
        printf("Error calculating elapsed time: %s\n", hipGetErrorString(err));
        return -1;
    }
    cout << "GPU Time: " << gpu_elapsed << "ms" << endl;
    hipMemcpy(h_img_blur_gpu, d_img_blur_gpu, img_size * sizeof(float), hipMemcpyDeviceToHost);
    showImg(h_img_blur_gpu, img_rows, img_cols, "Blurred Image GPU", 512, 512);
    
    // 比较图像
    for (int i = 0; i < img_size; i++)
    {
        if (abs(h_img_blur_cpu[i] - h_img_blur_gpu[i]) > 1e-3)
        {
            cout << "GPU Convolution is incorrect!" << endl;
            break;
        }
    }

    delete[] h_img;
    delete[] h_mask;
    delete[] h_img_blur_cpu;
    delete[] h_img_blur_gpu;

    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);

    return 0;
}