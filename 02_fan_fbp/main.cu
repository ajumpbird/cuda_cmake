#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <iostream>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <opencv2/opencv.hpp>
#include "tools.h"
#include "kernel.h"

using namespace std;
using namespace cv;

int main()
{
    // 设置几何参数
    Geo geo;
    geo.sod = 490.0f;
    geo.sdd = 880.0f;
    geo.detector_num = 2048;
    geo.detector_size = 0.2f;
    geo.views = 4000;
    geo.detector_length = geo.detector_num * geo.detector_size;
    geo.r = geo.sod * (geo.detector_length / 2) / sqrt((geo.detector_length / 2) * (geo.detector_length / 2) + geo.sdd * geo.sdd);
    geo.pixel_num = 2048;
    geo.pixel_size = 2 * geo.r / geo.pixel_num;
    geo.step_size = geo.pixel_size / 2;
    geo.padding_num = 2 * geo.detector_num;
    geo.da = 2 * M_PI / geo.views;

    // 记录开始时间
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // 读取原始图像
    const char *image_path = "data/img.dat";
    int img_size = geo.pixel_num * geo.pixel_num;
    float *h_img = new float[img_size];
    readFile(image_path, h_img, img_size);
    // showImg(h_img, geo.pixel_num, geo.pixel_num, "Original Image", 512, 512);

    // 创建纹理对象，将图像数据绑定到纹理对象
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipArray *cuArray;
    hipMallocArray(&cuArray, &channelDesc, geo.pixel_num, geo.pixel_num);
    hipTextureObject_t texObj = bindTexObj(cuArray, h_img, geo.pixel_num, geo.pixel_num);

    // sino 图
    int sino_size = geo.views * geo.detector_num;
    float *h_sino = new float[sino_size];
    float *d_sino;
    hipMalloc(&d_sino, sino_size * sizeof(float));

    // 生成 sinogram
    dim3 block(16, 16);
    dim3 grid((geo.detector_num + block.x - 1) / block.x, (geo.views + block.y - 1) / block.y);
    getSino<<<grid, block>>>(d_sino, geo, texObj);
    hipMemcpy(h_sino, d_sino, sino_size * sizeof(float), hipMemcpyDeviceToHost);
    // showImg(h_sino, geo.views, geo.detector_num, "Sinogram", 512, 512);

    // 得到填充加权 sino
    int pws_size = geo.views * geo.padding_num;
    float *d_pws;
    hipMalloc(&d_pws, pws_size * sizeof(float));
    dim3 block_pws(16, 16);
    dim3 grid_pws((geo.detector_num + block_pws.x - 1) / block_pws.x, geo.views, (geo.views + block_pws.y - 1) / block_pws.y);
    padWeightSino<<<grid_pws, block_pws>>>(d_pws, d_sino, geo);

    // 创建滤波器
    hipfftComplex *H;
    hipMalloc((void **)&H, (geo.padding_num / 2 + 1) * sizeof(hipfftComplex));
    getH(H, geo);

    // rec_img 图
    float *h_rec_img = new float[img_size];
    memset(h_rec_img, 0, img_size * sizeof(float));
    float *d_rec_img;
    hipMalloc(&d_rec_img, img_size * sizeof(float));
    hipMemset(d_rec_img, 0, img_size * sizeof(float));

    // 重建图像
    recImg(d_pws, H, d_rec_img, geo);

    // 记录结束时间
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "Elapsed Time: " << elapsedTime << "ms" << endl;

    hipMemcpy(h_rec_img, d_rec_img, img_size * sizeof(float), hipMemcpyDeviceToHost);
    showImg(h_rec_img, geo.pixel_num, geo.pixel_num, "Reconstructed Image", 512, 512);

    // 释放内存
    delete[] h_img;
    delete[] h_sino;
    delete[] h_rec_img;
    
    hipFree(d_sino);
    hipFree(d_pws);
    hipFree(H);
    hipFree(d_rec_img);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}